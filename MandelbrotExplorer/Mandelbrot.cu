
#include <hip/hip_runtime.h>
﻿extern "C"{
    __declspec( dllexport ) int* runMandelbrot(int _width, int _height, double scale, double offsetX, double offsetY, int iterations, int threshold);
    __declspec( dllexport ) void clearMem(int* ptr);
}

__global__ void mandelKernel(int* data, int _width, int _height, double scale, double offsetX, double offsetY, int iterations, int threshold){

    int strideY = gridDim.x;
    int strideX = blockDim.x;
    int startY = blockIdx.x;
    int startX = threadIdx.x;

    for(int y = startY; y < _height; y += strideY){
        for(int x = startX; x < _width; x += strideX){
            
            double cReal, cImaginary;
            
            int tempX = x;
            int tempY = y;
            
            tempX -= (_width / 2);
            tempY -= (_height / 2);
            
            cReal = (tempX / (_width / scale)) + offsetX;
            cImaginary = (tempY / (_width / scale)) + offsetY;
            
            double zReal = 0;
            double zImaginary = 0;
            
            bool escapes = false;
            int i = 0;
            while(i < iterations && !escapes){
                double oldZreal = zReal;
                zReal = ((zReal * zReal) - (zImaginary * zImaginary)) + cReal;
                zImaginary = (2 * oldZreal * zImaginary) + cImaginary;
                
                if((zReal * zReal)+(zImaginary * zImaginary) > threshold * threshold){
                    escapes = true;
                    data[(y * _width) + x] = i;
                }
                ++i;
            }
            if(!escapes){
                data[(y * _width) + x] = -1;
            }
        }
    }
    
    return;
}

int* runMandelbrot(int _width, int _height, double scale, double offsetX, double offsetY, int iterations, int threshold){

    int* host_data = new int[_width * _height];
    int* device_data;
    hipMalloc(&device_data, _width * _height * sizeof(int));

    mandelKernel<<<64, 1024>>>(device_data, _width, _height, scale, offsetX, offsetY, iterations, threshold);

    hipMemcpy(host_data, device_data, _width * _height * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_data);
    
    return host_data;
}

void clearMem(int* ptr){

    delete[] ptr;

}